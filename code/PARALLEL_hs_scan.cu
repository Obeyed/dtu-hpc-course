#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

// Performs one step of the hillis and steele algorithm for integers
__global__ void hs_kernel_global(unsigned int *d_out, unsigned int *d_in, int step, unsigned int SIZE) {
	// setting ID
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	// checking if out-of-bounds
	if (tid >= SIZE) return;
	// setting itself
	unsigned int val = d_in[tid];
	// finding the number to add, checking out-of-bounds
	unsigned int toAdd = (((tid - step) < 0) ? 0 : d_in[tid - step]);
	// setting output
	d_out[tid] = val + toAdd;
}

void hs_kernel_wrapper(unsigned int * d_out, unsigned int * d_in, unsigned int SIZE, unsigned int BYTES, unsigned int NUM_THREADS) {
	// initializing starting variables
	unsigned int NUM_BLOCKS = SIZE/NUM_THREADS + ((SIZE % NUM_THREADS)?1:0);
	int step = 1;
	// initializing and allocating an "intermediate" value so we don't have to change anything in d_in
	unsigned int * d_intermediate;
	hipMalloc((void **) &d_intermediate, BYTES);
	hipMemcpy(d_intermediate, d_in, BYTES, hipMemcpyDeviceToDevice);
	while(step<SIZE) // stops when step is larger than array size, happens at O(log2(SIZE))
	{
		hs_kernel_global<<<NUM_BLOCKS, NUM_THREADS>>>(d_out, d_intermediate, step, SIZE);
		hipMemcpy(d_intermediate, d_out, BYTES, hipMemcpyDeviceToDevice);
		step <<= 1; // double step size at each iteration

	}
	hipFree(d_intermediate);
}

int main(int argc, char **argv)
{
	std::ofstream myfile;
    myfile.open ("par_scan.csv");
	printf("Hillis and Steele ONLINE... \n");
	unsigned int times = 10;
	for (int rounds = 0; rounds<30; rounds++)
	{
		// defining vars
		unsigned int NUM_THREADS = 1<<10;
		unsigned int SIZE = 1<<rounds;
		unsigned int BYTES = SIZE * sizeof(unsigned int);
		printf("num: %d\n", rounds);		
		printf("SIZE: %d\n", SIZE);

		// setting host in
		unsigned int * h_in  = (unsigned int *)malloc(BYTES); // allocates to memory
		unsigned int * h_out = (unsigned int *)malloc(BYTES);
		for(unsigned int i = 0; i < SIZE; i++){h_in[i] = 1;}

		// setting device pointers
		unsigned int * d_in;
		unsigned int * d_out;

		// allocate GPU memory
		hipMalloc((void **) &d_in, BYTES);
		hipMalloc((void **) &d_out, BYTES);

		// transfer arrays to GPU
		hipMemcpy(d_in, h_in, BYTES, hipMemcpyHostToDevice);

		// setting up time
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		// kernel time!!!
		hipEventRecord(start, 0);
		for (unsigned int i = 0; i < times; i++)
	    {
	    	hs_kernel_wrapper(d_out, d_in, SIZE, BYTES, NUM_THREADS);
	    }
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		// calculating time
		float elapsedTime;
	    hipEventElapsedTime(&elapsedTime, start, stop);    
	    elapsedTime = elapsedTime / ((float) times);

		// back to host
		hipMemcpy(h_out, d_out, BYTES, hipMemcpyDeviceToHost);

		printf("average time elapsed: %f\n", elapsedTime);

		// free GPU memory allocation
		hipFree(d_in);
		hipFree(d_out);
        myfile << elapsedTime << ",";
	}
	myfile.close();
	return 0;
}
