#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Performs one step of the hillis and steele algorithm for integers
__global__ void hs_kernel_global(int *d_out, int *d_in, int step, int SIZE) {
	// setting ID
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	// checking if out-of-bounds
	if (tid >= SIZE) return;
	// setting itself
	int val = d_in[tid];
	// finding the number to add, checking out-of-bounds
	int toAdd = (((tid - step) < 0) ? 0 : d_in[tid - step]);
	// setting output
	d_out[tid] = val + toAdd;
}

void hs_kernel_wrapper(int * d_out, int * d_in, int SIZE, unsigned int BYTES, int NUM_THREADS) {
	// initializing starting variables
	int NUM_BLOCKS = SIZE/NUM_THREADS + ((SIZE % NUM_THREADS)?1:0);
	// initializing and allocating an "intermediate" value so we don't have to change anything in d_in
	int *d_intermediate;
	hipMalloc((void **) &d_intermediate, BYTES);
	hipMemcpy(d_intermediate, d_in, BYTES, hipMemcpyDeviceToDevice);

  // stops when step is larger than array size, happens at O(log2(SIZE))
  for (int step = 1; step < SIZE; step <<= 1) {
		hs_kernel_global<<<NUM_BLOCKS, NUM_THREADS>>>(d_out, d_intermediate, step, SIZE);
		hipMemcpy(d_intermediate, d_out, BYTES, hipMemcpyDeviceToDevice);
	}
	hipFree(d_intermediate);
}

int main(int argc, char **argv) {
  int NUM_THREADS = 1 << 10,
      SIZE,
      TIMES = 1;
  unsigned int BYTES;
  int *h_in, *h_out,
      *d_in, *d_out;
	for (int rounds = 29; rounds < 30; rounds++) {
		// defining vars
    SIZE  = 1 << rounds; 
    BYTES = SIZE * sizeof(int);

		// setting host memory
		h_in  = (int *)malloc(BYTES); 
		h_out = (int *)malloc(BYTES);

		for(int i = 0; i < SIZE; i++)
      h_in[i] = 1;

		// allocate GPU memory
		hipMalloc((void **) &d_in, BYTES);
		hipMalloc((void **) &d_out, BYTES);

		// transfer arrays to GPU
		hipMemcpy(d_in, h_in, BYTES, hipMemcpyHostToDevice);

		// kernel time!!!
		for (int i = 0; i < TIMES; i++)
	    hs_kernel_wrapper(d_out, d_in, SIZE, BYTES, NUM_THREADS);

		// back to host
		hipMemcpy(h_out, d_out, BYTES, hipMemcpyDeviceToHost);

		// free GPU memory allocation
		hipFree(d_in);
		hipFree(d_out);
	}

  for (int i = 0; i < 5; i++)
    printf("%d ", h_out[i]);

  printf(" -- ");

  for (int i = SIZE - 5; i < SIZE; i++)
    printf("%d ", h_out[i]);

  printf("\n");

	return 0;
}
