#include "hip/hip_runtime.h"
/**
    High Performance Computing (special course)
    radix_sort.cu
    Location: Technical University of Denmark
    Purpose: Uses GPU to sort series of unsigned integers using Radix Sort

    @author Elias Obeid
    @author Alexander Johansen
    @version 1.0 16/01/2016
*/

#include "radix_sort.h"

/**
    Populates array with 1/0 depending on Least Significant Bit is set.
    If LSB is 0 then index is set to 1, otherwise 0.

    @param d_predicate  Output array to be filled with values (predicates)
    @param d_val_src    Values to run through
    @param numElems     Number of elements in arrays
    @param i            Used to calculate how much to shift to find the correct LSB
*/
__global__ 
void predicate_kernel(unsigned int* const, 
                      const unsigned int* const,
                      const size_t,
                      const unsigned int);
/**
    Performs one iteration of Hillis and Steele scan.
    Inclusive sum scan.

    @param d_out    Output array with summed values
    @param d_in     Values to sum
    @param step     Amount to look back in d_in
    @param numElems Number of elements in arrays
*/
__global__
void inclusive_sum_scan_kernel(unsigned int* const,
                               const unsigned int* const,
                               const int,
                               const size_t);
/**
    Shifts all elements to the right.
    Sets first index to 0.

    @param d_out    Output array
    @param d_in     Array to be shifted
    @param numElems Number of elements in arrays
*/
__global__
void right_shift_array_kernel(unsigned int* const,
                              const unsigned int* const,
                              const size_t);
/**
    Toggle array with values 1 and 0.

    @param d_out        Array with toggled values
    @param d_predicate  Array with initial values
    @param numElems     Number of elements in arrays
*/
__global__
void toggle_predicate_kernel(unsigned int* const, 
                             const unsigned int* const,
                             const size_t);
/**
    Adds an offset to the given array's values.

    @param d_out      Input/Output array -- values will be added to offset
    @param shift      Array with one element -- the offset to add
    @param numElems   Number of elements in arrays
*/
__global__
void add_splitter_map_kernel(unsigned int* const,
                             const unsigned int* const, 
                             const size_t);
/**
    Runs log_2(BLOCK_SIZE) iterations of the reduce.
    Computes the sum of elements in d_in

    @param d_out     Output array
    @param d_in      Input array with values
    @param numElems  Number of elements in arrays
*/
__global__ 
void reduce_kernel(unsigned int* const,
                   unsigned int* const,
                   const size_t);
/**
    Maps values from d_in to d_out according to scatter addresses in d_sum_scan_0 or d_sum_scan_1.

    @param d_out        Output array
    @param d_in         Input array with values
    @param d_predicate  Contains whether or not given value's LSB is 0
    @param d_sum_scan_0 Scatter address for values with LSB 0
    @param d_sum_scan_1 Scatter address for values with LSB 1
    @param numElems     Number of elements in arrays
*/
__global__
void map_kernel(unsigned int* const,
                const unsigned int* const,
                const unsigned int* const,
                const unsigned int* const,
                const unsigned int* const,
                const size_t);
/**
    Calls reduce kernel to compute reduction.
    Runs log_(BLOCK_SIZE)(numElems) times.

    @param d_out      Output array
    @param d_in       Input array with values
    @param numElems   Number of elements in arrays
    @param block_size Number of threads per block
*/
void reduce_wrapper(unsigned int* const,
                    unsigned int* const,
                    size_t,
                    int);
/**
    Computes an exclusive sum scan of scatter addresses for the given predicate array.

    @param d_out            Output array with scatter addresses
    @param d_predicate      Input array with predicates to be summed
    @param d_predicate_tmp  Temporary array so we do not change d_predicate
    @param d_sum_scan       Inclusive sum scan
    @param ARRAY_BYTES      Number of bytes for arrays
    @param numElems         Number of elements in arrays
    @param GRID_SIZE        Number of blocks in one grid
    @param BLOCK_SIZE       Number of threads in one block
*/
void exclusive_sum_scan(unsigned int* const,
                        const unsigned int* const,
                        unsigned int* const,
                        unsigned int* const,
                        const unsigned int,
                        const size_t,
                        const int,
                        const int);
/**
    Computes an exclusive sum scan of scatter addresses for the given predicate array.

    @param h_input  Input values to be sorted (unsigned int)
    @param numElems Number of elements in array
    @return Pointer to sorted array
*/
unsigned int* radix_sort(unsigned int*,
                         const size_t);
// Populates array with 1/0 depending on Least Significant Bit is set.
__global__
void predicate_kernel(unsigned int* const d_predicate,
                      const unsigned int* const d_val_src,
                      const size_t numElems,
                      const unsigned int i) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  d_predicate[mid] = (int)(((d_val_src[mid] & (1 << i)) >> i) == 0);
}

// Performs one iteration of Hillis and Steele scan.
__global__
void inclusive_sum_scan_kernel(unsigned int* const d_out,
                               const unsigned int* const d_in,
                               const int step,
                               const size_t numElems) {
  const int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

	int toAdd = (((mid - step) < 0) ? 0 : d_in[mid - step]);
  d_out[mid] = d_in[mid] + toAdd;
}

// Shifts all elements to the right. Sets first index to 0.
__global__
void right_shift_array_kernel(unsigned int* const d_out,
                       const unsigned int* const d_in,
                       const size_t numElems) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  d_out[mid] = (mid == 0) ? 0 : d_in[mid - 1];
}

// Toggle array with values 1 and 0.
__global__
void toggle_predicate_kernel(unsigned int* const d_out, 
                             const unsigned int* const d_predicate,
                             const size_t numElems) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  d_out[mid] = ((d_predicate[mid]) ? 0 : 1);
}

// Adds an offset to the given array's values.
__global__
void add_splitter_map_kernel(unsigned int* const d_out,
                             const unsigned int* const shift, 
                             const size_t numElems) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  d_out[mid] += shift[0];
}

// Computes the sum of elements in d_in
__global__ 
void reduce_kernel(unsigned int* const d_out,
                   unsigned int* const d_in,
                   const size_t numElems) {
  unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;

  for (unsigned int s = blockDim.x / 2; s > 0; s >>=1) {
    if ((tid < s) && ((pos + s) < numElems))
      d_in[pos] = d_in[pos] + d_in[pos + s];
    __syncthreads();
  }

  // only thread 0 writes result, as thread
  if ((tid == 0) && (pos < numElems))
    d_out[blockIdx.x] = d_in[pos];
}

// Maps values from d_in to d_out according to scatter addresses in d_sum_scan_0 or d_sum_scan_1.
__global__
void map_kernel(unsigned int* const d_out,
                const unsigned int* const d_in,
                const unsigned int* const d_predicate,
                const unsigned int* const d_sum_scan_0,
                const unsigned int* const d_sum_scan_1,
                const size_t numElems) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  const unsigned int pos = ((d_predicate[mid]) ? d_sum_scan_0[mid] : d_sum_scan_1[mid]);
  d_out[pos] = d_in[mid];
}

// Calls reduce kernel to compute reduction.
void reduce_wrapper(unsigned int* const d_out,
                    unsigned int* const d_in,
                    size_t numElems,
                    int block_size) {
  unsigned int grid_size = numElems / block_size + 1;

  unsigned int* d_tmp;
  checkCudaErrors(hipMalloc(&d_tmp, sizeof(unsigned int) * grid_size));
  checkCudaErrors(hipMemset(d_tmp, 0, sizeof(unsigned int) * grid_size));

  unsigned int prev_grid_size;
  unsigned int remainder = 0;
  // recursively solving, will run approximately log base block_size times.
  do {
    reduce_kernel<<<grid_size, block_size>>>(d_tmp, d_in, numElems);

    remainder = numElems % block_size;
    numElems  = numElems / block_size + remainder;

    // updating input to intermediate
    checkCudaErrors(hipMemcpy(d_in, d_tmp, sizeof(int) * grid_size, hipMemcpyDeviceToDevice));

    // Updating grid_size to reflect how many blocks we now want to compute on
    prev_grid_size = grid_size;
    grid_size = numElems / block_size + 1;      

    // updating intermediate
    checkCudaErrors(hipFree(d_tmp));
    checkCudaErrors(hipMalloc(&d_tmp, sizeof(int) * grid_size));
  } while(numElems > block_size);

  // computing rest
  reduce_kernel<<<1, numElems>>>(d_out, d_in, prev_grid_size);
}

// Computes an exclusive sum scan of scatter addresses for the given predicate array.
void exclusive_sum_scan(unsigned int* const d_out,
                        const unsigned int* const d_predicate,
                        unsigned int* const d_predicate_tmp,
                        unsigned int* const d_sum_scan,
                        const unsigned int ARRAY_BYTES,
                        const size_t numElems,
                        const int GRID_SIZE,
                        const int BLOCK_SIZE) {
  // copy predicate values to new array
  checkCudaErrors(hipMemcpy(d_predicate_tmp, d_predicate, ARRAY_BYTES, hipMemcpyDeviceToDevice));
  // set all elements to zero 
  checkCudaErrors(hipMemset(d_sum_scan, 0, ARRAY_BYTES));

  // sum scan call
  for (unsigned int step = 1; step < numElems; step *= 2) {
    inclusive_sum_scan_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_sum_scan, d_predicate_tmp, step, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpy(d_predicate_tmp, d_sum_scan, ARRAY_BYTES, hipMemcpyDeviceToDevice));
  }

  // shift to get exclusive scan
  checkCudaErrors(hipMemcpy(d_out, d_sum_scan, ARRAY_BYTES, hipMemcpyDeviceToDevice));
  right_shift_array_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_out, d_sum_scan, numElems);
}

// Computes an exclusive sum scan of scatter addresses for the given predicate array.
unsigned int* radix_sort(unsigned int* h_input,
                         const size_t numElems) {
  const int BLOCK_SIZE  = 512;
  const int GRID_SIZE   = numElems / BLOCK_SIZE + 1;
  const unsigned int ARRAY_BYTES = sizeof(unsigned int) * numElems;
  const unsigned int BITS_PER_BYTE = 8;

  // host memory
  unsigned int* const h_output = new unsigned int[numElems];

  // device memory
  unsigned int *d_val_src, *d_predicate, *d_sum_scan, *d_predicate_tmp, *d_sum_scan_0, *d_sum_scan_1, *d_predicate_toggle, *d_reduce, *d_map;
  checkCudaErrors(hipMalloc((void **) &d_val_src,          ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_map,              ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_predicate,        ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_predicate_tmp,    ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_predicate_toggle, ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_sum_scan,         ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_sum_scan_0,       ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_sum_scan_1,       ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_reduce, sizeof(unsigned int)));

  // copy host array to device
  checkCudaErrors(hipMemcpy(d_val_src, h_input, ARRAY_BYTES, hipMemcpyHostToDevice));

  for (unsigned int i = 0; i < (BITS_PER_BYTE * sizeof(unsigned int)); i++) {
    // predicate is that LSB is 0
    predicate_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_predicate, d_val_src, numElems, i);

    // calculate scatter addresses from predicates
    exclusive_sum_scan(d_sum_scan_0, d_predicate, d_predicate_tmp, d_sum_scan, ARRAY_BYTES, numElems, GRID_SIZE, BLOCK_SIZE);

    // copy contents of predicate, so we do not change its content
    checkCudaErrors(hipMemcpy(d_predicate_tmp, d_predicate, ARRAY_BYTES, hipMemcpyDeviceToDevice));

    // calculate how many elements had predicate equal to 1
    reduce_wrapper(d_reduce, d_predicate_tmp, numElems, BLOCK_SIZE);

    // toggle predicate values, so we can compute scatter addresses for toggled predicates
    toggle_predicate_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_predicate_toggle, d_predicate, numElems);
    // so we now have addresses for elements where LSB is equal to 1
    exclusive_sum_scan(d_sum_scan_1, d_predicate_toggle, d_predicate_tmp, d_sum_scan, ARRAY_BYTES, numElems, GRID_SIZE, BLOCK_SIZE);
    // shift scatter addresses according to amount of elements that had LSB equal to 0
    add_splitter_map_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_sum_scan_1, d_reduce, numElems);

    // move elements accordingly
    map_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_map, d_val_src, d_predicate, d_sum_scan_0, d_sum_scan_1, numElems);

    // swap pointers, instead of moving elements
    std::swap(d_val_src, d_map);
  }

  // copy contents back
  checkCudaErrors(hipMemcpy(h_output, d_val_src, ARRAY_BYTES, hipMemcpyDeviceToHost));

  return h_output;
}
