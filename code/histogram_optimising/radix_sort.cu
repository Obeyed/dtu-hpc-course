#include "hip/hip_runtime.h"
/**
    High Performance Computing (special course)
    radix_sort.cu
    Location: Technical University of Denmark
    Purpose: Uses GPU to sort series of unsigned integers using Radix Sort
    EDIT: This radix sort has been modified to take multiple arrays.
          We need to sort three arrays by some specific array.
          The radix sort is modified accordingly.

    @author Elias Obeid
    @author Alexander Johansen
    @version 1.0 16/01/2016
*/

#include "radix_sort.h"

/**
    Populates array with 1/0 depending on Least Significant Bit is set.
    If LSB is 0 then index is set to 1, otherwise 0.

    @param d_predicate  Output array to be filled with values (predicates)
    @param d_sort_by    Values to run through
    @param NUM_ELEMS     Number of elements in arrays
    @param i            Used to calculate how much to shift to find the correct LSB
*/
__global__ 
void predicate_kernel(unsigned int* const, 
                      const unsigned int* const,
                      const size_t,
                      const unsigned int);
/**
    Performs one iteration of Hillis and Steele scan.
    Inclusive sum scan.

    @param d_out    Output array with summed values
    @param d_in     Values to sum
    @param step     Amount to look back in d_in
    @param NUM_ELEMS Number of elements in arrays
*/
__global__
void inclusive_sum_scan_kernel(unsigned int* const,
                               const unsigned int* const,
                               const int,
                               const size_t);
/**
    Shifts all elements to the right.
    Sets first index to 0.

    @param d_out    Output array
    @param d_in     Array to be shifted
    @param NUM_ELEMS Number of elements in arrays
*/
__global__
void right_shift_array_kernel(unsigned int* const,
                              const unsigned int* const,
                              const size_t);
/**
    Toggle array with values 1 and 0.

    @param d_out        Array with toggled values
    @param d_predicate  Array with initial values
    @param NUM_ELEMS     Number of elements in arrays
*/
__global__
void toggle_predicate_kernel(unsigned int* const, 
                             const unsigned int* const,
                             const size_t);
/**
    Adds an offset to the given array's values.

    @param d_out      Input/Output array -- values will be added to offset
    @param shift      Array with one element -- the offset to add
    @param NUM_ELEMS   Number of elements in arrays
*/
__global__
void add_splitter_map_kernel(unsigned int* const,
                             const unsigned int* const, 
                             const size_t);
/**
    Runs log_2(BLOCK_SIZE) iterations of the reduce.
    Computes the sum of elements in d_in

    @param d_out     Output array
    @param d_in      Input array with values
    @param NUM_ELEMS  Number of elements in arrays
*/
__global__ 
void reduce_kernel(unsigned int* const,
                   unsigned int* const,
                   const size_t);
/**
    Maps values from d_in to d_out according to scatter addresses in d_sum_scan_0 or d_sum_scan_1.

    @param d_out        Output array
    @param d_in         Input array with values
    @param d_predicate  Contains whether or not given value's LSB is 0
    @param d_sum_scan_0 Scatter address for values with LSB 0
    @param d_sum_scan_1 Scatter address for values with LSB 1
    @param NUM_ELEMS     Number of elements in arrays
*/
__global__
void map_kernel(unsigned int* const,
                unsigned int* const,
                unsigned int* const,
                const unsigned int* const,
                const unsigned int* const,
                const unsigned int* const,
                const unsigned int* const,
                const unsigned int* const,
                const unsigned int* const,
                const size_t);
/**
    Calls reduce kernel to compute reduction.
    Runs log_(BLOCK_SIZE)(num_elems) times.

    @param d_out      Output array
    @param d_in       Input array with values
    @param num_elems   Number of elements in arrays
    @param block_size Number of threads per block
*/
void reduce_wrapper(unsigned int* const,
                    unsigned int* const,
                    size_t,
                    int,
                    GpuTimer,
                    float&);
/**
    Computes an exclusive sum scan of scatter addresses for the given predicate array.

    @param d_out            Output array with scatter addresses
    @param d_predicate      Input array with predicates to be summed
    @param d_predicate_tmp  Temporary array so we do not change d_predicate
    @param d_sum_scan       Inclusive sum scan
    @param ARRAY_BYTES      Number of bytes for arrays
    @param NUM_ELEMS         Number of elements in arrays
    @param GRID_SIZE        Number of blocks in one grid
    @param BLOCK_SIZE       Number of threads in one block
*/
void exclusive_sum_scan(unsigned int* const,
                        const unsigned int* const,
                        unsigned int* const,
                        unsigned int* const,
                        const unsigned int,
                        const size_t,
                        const int,
                        const int,
                        GpuTimer,
                        float&);
/**
    Sort values using radix sort.

    @param h_input  Input values to be sorted (unsigned int)
    @param NUM_ELEMS Number of elements in array
    @return Pointer to sorted array
*/
unsigned int* radix_sort(unsigned int*,
                         const size_t);
// Populates array with 1/0 depending on Least Significant Bit is set.
__global__
void predicate_kernel(unsigned int* const d_predicate,
                      const unsigned int* const d_sort_by,
                      const size_t NUM_ELEMS,
                      const unsigned int i) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= NUM_ELEMS) return;

  d_predicate[mid] = (int)(((d_sort_by[mid] & (1 << i)) >> i) == 0);
}

// Performs one iteration of Hillis and Steele scan.
__global__
void inclusive_sum_scan_kernel(unsigned int* const d_out,
                               const unsigned int* const d_in,
                               const int step,
                               const size_t NUM_ELEMS) {
  const int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= NUM_ELEMS) return;

	int toAdd = (((mid - step) < 0) ? 0 : d_in[mid - step]);
  d_out[mid] = d_in[mid] + toAdd;
}

// Shifts all elements to the right. Sets first index to 0.
__global__
void right_shift_array_kernel(unsigned int* const d_out,
                       const unsigned int* const d_in,
                       const size_t NUM_ELEMS) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= NUM_ELEMS) return;

  d_out[mid] = (mid == 0) ? 0 : d_in[mid - 1];
}

// Toggle array with values 1 and 0.
__global__
void toggle_predicate_kernel(unsigned int* const d_out, 
                             const unsigned int* const d_predicate,
                             const size_t NUM_ELEMS) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= NUM_ELEMS) return;

  d_out[mid] = ((d_predicate[mid]) ? 0 : 1);
}

// Adds an offset to the given array's values.
__global__
void add_splitter_map_kernel(unsigned int* const d_out,
                             const unsigned int* const shift, 
                             const size_t NUM_ELEMS) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= NUM_ELEMS) return;

  d_out[mid] += shift[0];
}

// Computes the sum of elements in d_in
__global__ 
void reduce_kernel(unsigned int* const d_out,
                   unsigned int* const d_in,
                   const size_t NUM_ELEMS) {
  unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;

  for (unsigned int s = blockDim.x / 2; s > 0; s >>=1) {
    if ((tid < s) && ((pos + s) < NUM_ELEMS))
      d_in[pos] = d_in[pos] + d_in[pos + s];
    __syncthreads();
  }

  // only thread 0 writes result, as thread
  if ((tid == 0) && (pos < NUM_ELEMS))
    d_out[blockIdx.x] = d_in[pos];
}

// Maps values from d_in to d_out according to scatter addresses in d_sum_scan_0 or d_sum_scan_1.
__global__
void map_kernel(unsigned int* const d_out_coarse,
                unsigned int* const d_out_bin,
                unsigned int* const d_out_val,
                const unsigned int* const d_in_coarse,
                const unsigned int* const d_in_bin,
                const unsigned int* const d_in_val,
                const unsigned int* const d_predicate,
                const unsigned int* const d_sum_scan_0,
                const unsigned int* const d_sum_scan_1,
                const size_t NUM_ELEMS) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= NUM_ELEMS) return;

  const unsigned int pos = ((d_predicate[mid]) ? d_sum_scan_0[mid] : d_sum_scan_1[mid]);
  // EDIT: MOVE ACCORDINGLY FOR ALL ARRAYS 
  d_out_val[pos]    = d_in_val[mid];
  d_out_bin[pos]    = d_in_bin[mid];
  d_out_coarse[pos] = d_in_coarse[mid];
}

// Calls reduce kernel to compute reduction.
void reduce_wrapper(unsigned int* const d_out,
                    unsigned int* const d_in,
                    size_t num_elems,
                    int block_size,
                    GpuTimer timer,
                    float& elapsed) {
  unsigned int grid_size = num_elems / block_size + 1;

  unsigned int* d_tmp;
  checkCudaErrors(hipMalloc(&d_tmp, sizeof(unsigned int) * grid_size));
  checkCudaErrors(hipMemset(d_tmp, 0, sizeof(unsigned int) * grid_size));

  unsigned int prev_grid_size;
  unsigned int remainder = 0;
  // recursively solving, will run approximately log base block_size times.
  do {
    timer.Start();
    reduce_kernel<<<grid_size, block_size>>>(d_tmp, d_in, num_elems);
    timer.Stop();
    elapsed += timer.Elapsed();

    remainder = num_elems % block_size;
    num_elems  = num_elems / block_size + remainder;

    // updating input to intermediate
    checkCudaErrors(hipMemcpy(d_in, d_tmp, sizeof(int) * grid_size, hipMemcpyDeviceToDevice));

    // Updating grid_size to reflect how many blocks we now want to compute on
    prev_grid_size = grid_size;
    grid_size = num_elems / block_size + 1;      

    // updating intermediate
    checkCudaErrors(hipFree(d_tmp));
    checkCudaErrors(hipMalloc(&d_tmp, sizeof(int) * grid_size));
  } while(num_elems > block_size);

  // computing rest
  timer.Start();
  reduce_kernel<<<1, num_elems>>>(d_out, d_in, prev_grid_size);
  timer.Stop();
  elapsed += timer.Elapsed();
}

// Computes an exclusive sum scan of scatter addresses for the given predicate array.
void exclusive_sum_scan(unsigned int* const d_out,
                        const unsigned int* const d_predicate,
                        unsigned int* const d_predicate_tmp,
                        unsigned int* const d_sum_scan,
                        const unsigned int ARRAY_BYTES,
                        const size_t NUM_ELEMS,
                        const int GRID_SIZE,
                        const int BLOCK_SIZE,
                        GpuTimer timer,
                        float& elapsed) {
  // copy predicate values to new array
  checkCudaErrors(hipMemcpy(d_predicate_tmp, d_predicate, ARRAY_BYTES, hipMemcpyDeviceToDevice));
  // set all elements to zero 
  checkCudaErrors(hipMemset(d_sum_scan, 0, ARRAY_BYTES));

  // sum scan call
  for (unsigned int step = 1; step < NUM_ELEMS; step *= 2) {
    printf("step: %u, elapsed: %f\n", step, elapsed);
    timer.Start();
    inclusive_sum_scan_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_sum_scan, d_predicate_tmp, step, NUM_ELEMS);
    timer.Stop();
    elapsed += timer.Elapsed();
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpy(d_predicate_tmp, d_sum_scan, ARRAY_BYTES, hipMemcpyDeviceToDevice));
  }

  // shift to get exclusive scan
  checkCudaErrors(hipMemcpy(d_out, d_sum_scan, ARRAY_BYTES, hipMemcpyDeviceToDevice));
  timer.Start();
  right_shift_array_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_out, d_sum_scan, NUM_ELEMS);
  timer.Stop();
  elapsed += timer.Elapsed();
}

// Sort values using radix sort
// EDIT: sort by first array in h_to_be_sorted
unsigned int** radix_sort(float& elapsed,
                          unsigned int** h_to_be_sorted,
                          const size_t NUM_ARRAYS_TO_SORT,
                          const size_t NUM_ELEMS) {
  GpuTimer timer;

  const int BLOCK_SIZE  = 1024;
  const int GRID_SIZE   = NUM_ELEMS / BLOCK_SIZE + 1;
  const unsigned int ARRAY_BYTES = sizeof(unsigned int) * NUM_ELEMS;
  const unsigned int BITS_PER_BYTE = 8;

  // host memory
  unsigned int** h_output = new unsigned int*[NUM_ARRAYS_TO_SORT];
  unsigned int* h_out_coarse = new unsigned int[NUM_ELEMS];
  unsigned int* h_out_bin = new unsigned int[NUM_ELEMS];
  unsigned int* h_out_val = new unsigned int[NUM_ELEMS];

  // device memory
  unsigned int *d_in_bin, *d_in_val, *d_sort_by, *d_map_coarse, *d_map_val, 
               *d_map_bin, *d_predicate, *d_sum_scan, *d_predicate_tmp, 
               *d_sum_scan_0, *d_sum_scan_1, *d_predicate_toggle, *d_reduce;
  checkCudaErrors(hipMalloc((void **) &d_sort_by,          ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_in_bin,           ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_in_val,           ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_map_coarse,       ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_map_val,          ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_map_bin,          ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_predicate,        ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_predicate_tmp,    ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_predicate_toggle, ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_sum_scan,         ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_sum_scan_0,       ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_sum_scan_1,       ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_reduce, sizeof(unsigned int)));

  // copy host array to device
  checkCudaErrors(hipMemcpy(d_sort_by, h_to_be_sorted[0], ARRAY_BYTES, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_in_bin,  h_to_be_sorted[1], ARRAY_BYTES, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_in_val,  h_to_be_sorted[2], ARRAY_BYTES, hipMemcpyHostToDevice));

  for (unsigned int i = 0; i < (BITS_PER_BYTE * sizeof(unsigned int)); i++) {
    // predicate is that LSB is 0
    timer.Start();
    predicate_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_predicate, d_sort_by, NUM_ELEMS, i);
    timer.Stop();
    elapsed += timer.Elapsed();

    // calculate scatter addresses from predicates
    printf("first\n");
    exclusive_sum_scan(d_sum_scan_0, d_predicate, d_predicate_tmp, d_sum_scan, ARRAY_BYTES, NUM_ELEMS, GRID_SIZE, BLOCK_SIZE, timer, elapsed);

    // copy contents of predicate, so we do not change its content
    checkCudaErrors(hipMemcpy(d_predicate_tmp, d_predicate, ARRAY_BYTES, hipMemcpyDeviceToDevice));

    // calculate how many elements had predicate equal to 1
    reduce_wrapper(d_reduce, d_predicate_tmp, NUM_ELEMS, BLOCK_SIZE, timer, elapsed);

    // toggle predicate values, so we can compute scatter addresses for toggled predicates
    timer.Start();
    toggle_predicate_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_predicate_toggle, d_predicate, NUM_ELEMS);
    timer.Stop();
    elapsed += timer.Elapsed();
    // so we now have addresses for elements where LSB is equal to 1
    printf("second\n");
    exclusive_sum_scan(d_sum_scan_1, d_predicate_toggle, d_predicate_tmp, d_sum_scan, ARRAY_BYTES, NUM_ELEMS, GRID_SIZE, BLOCK_SIZE, timer, elapsed);
    // shift scatter addresses according to amount of elements that had LSB equal to 0
    timer.Start();
    add_splitter_map_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_sum_scan_1, d_reduce, NUM_ELEMS);
    timer.Stop();
    elapsed += timer.Elapsed();

    // move elements accordingly
    timer.Start();
    map_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_map_coarse, d_map_bin, d_map_val, 
                                         d_sort_by, d_in_bin, d_in_val, 
                                         d_predicate, d_sum_scan_0, d_sum_scan_1, NUM_ELEMS);
    timer.Stop();
    elapsed += timer.Elapsed();

    // swap pointers, instead of moving elements
    std::swap(d_sort_by, d_map_coarse);
    std::swap(d_in_bin, d_map_bin);
    std::swap(d_in_val, d_map_val);
  }

  // copy contents back
  checkCudaErrors(hipMemcpy(h_out_coarse, d_sort_by, ARRAY_BYTES, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_out_bin, d_map_bin, ARRAY_BYTES, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_out_val, d_map_val, ARRAY_BYTES, hipMemcpyDeviceToHost));

  h_output[0] = h_out_coarse;
  h_output[1] = h_out_bin;
  h_output[2] = h_out_val;

  return h_output;
}
