#include "hip/hip_runtime.h"
#include "par_scan.h"

// Performs one step of the Hillis and Steele algorithm for unsigned integers
__global__ 
void scan_kernel(unsigned int* const d_out, 
                 unsigned int* const d_in, 
                 int step, 
                 size_t SIZE) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= SIZE) return;

  int val = d_in[tid];
  int toAdd = (((tid - step) < 0) ? 0 : d_in[tid - step]);
  d_out[tid] = val + toAdd;
}

void scan_wrapper(unsigned int* const d_out, 
                  unsigned int* const d_in, 
                  const size_t SIZE, 
                  const unsigned int BYTES,
                  const unsigned int BLOCK_SIZE) {
  int GRID_SIZE = SIZE/BLOCK_SIZE + 1;

  // device memory
  unsigned int *d_tmp;
  hipMalloc((void **) &d_tmp, BYTES);
  hipMemcpy(d_tmp, d_in, BYTES, hipMemcpyDeviceToDevice);

  // stops when step is larger than array size, happens at O(log2(SIZE))
  for (int step = 1; step < SIZE; step <<= 1) {
    scan_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_out, d_tmp, step, SIZE);
    hipMemcpy(d_tmp, d_out, BYTES, hipMemcpyDeviceToDevice);
  }
  hipFree(d_tmp);
}

void par_scan(unsigned int* const h_out, 
              unsigned int* const h_in,
              const size_t NUM_ELEMS) {
  const int BLOCK_SIZE  = 512;
  const unsigned int ARRAY_BYTES = sizeof(unsigned int) * NUM_ELEMS;

  // device memory
  unsigned int *d_in, *d_out;
  checkCudaErrors(hipMalloc((void **) &d_in,  ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_out, ARRAY_BYTES));

  // Transfer the arrays to the GPU
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
  // run kernel
  scan_wrapper(d_out, d_in, NUM_ELEMS, ARRAY_BYTES, BLOCK_SIZE);
  // copy values to host
  hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
  // free device memory
  hipFree(d_in); hipFree(d_out);
}
