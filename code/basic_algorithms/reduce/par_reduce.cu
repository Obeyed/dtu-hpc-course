#include "hip/hip_runtime.h"
#include "par_reduce.h"

// Computes the sum of elements in d_in in global memory
__global__ 
void global_reduce_kernel(unsigned int* const d_out,
                          unsigned int* const d_in,
                          const size_t NUM_ELEMS) {
  unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;

  for (unsigned int s = blockDim.x / 2; s > 0; s >>=1) {
    if ((tid < s) && ((pos + s) < NUM_ELEMS))
      d_in[pos] = d_in[pos] + d_in[pos + s];
    __syncthreads();
  }

  // only thread 0 writes result, as thread
  if ((tid == 0) && (pos < NUM_ELEMS))
    d_out[blockIdx.x] = d_in[pos];
}

// Computes the sum of elements in d_in in shared memory
__global__ 
void shared_reduce_kernel(unsigned int* const d_out,
                          unsigned int* const d_in,
                          const size_t NUM_ELEMS) {
  unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;

  extern __shared__ unsigned int* sdata;  // allocate shared memory
  sdata[tid] = d_in[pos];                 // each thread loads global to shared
  __syncthreads();                        // make sure all threads are done

  for (unsigned int s = blockDim.x / 2; s > 0; s >>=1) {
    if ((tid < s) && ((pos + s) < NUM_ELEMS))
      sdata[tid] +=  sdata[tid + s];      // perform operations on shared memory
    __syncthreads();
  }

  // only thread 0 writes result, as thread
  if ((tid == 0) && (pos < NUM_ELEMS))
    d_out[blockIdx.x] = sdata[0];         // copy shared back to global
}

// Calls reduce kernel to compute reduction.
void reduce_wrapper(unsigned int* const d_out,
                    unsigned int* const d_in,
                    size_t num_elems,
                    const int BLOCK_SIZE) {
  unsigned int grid_size = num_elems / BLOCK_SIZE + 1;
  const unsigned int SMEM = BLOCK_SIZE * sizeof(unsigned int);

  unsigned int* d_tmp;
  checkCudaErrors(hipMalloc(&d_tmp, sizeof(unsigned int) * grid_size));
  checkCudaErrors(hipMemset(d_tmp, 0, sizeof(unsigned int) * grid_size));

  unsigned int prev_grid_size;
  unsigned int remainder = 0;
  // recursively solving, will run approximately log base BLOCK_SIZE times.
  do {
    //global_reduce_kernel<<<grid_size, BLOCK_SIZE>>>(d_tmp, d_in, num_elems);
    shared_reduce_kernel<<<grid_size, BLOCK_SIZE, SMEM>>>(d_tmp, d_in, num_elems);

    remainder = num_elems % BLOCK_SIZE;
    num_elems  = num_elems / BLOCK_SIZE + remainder;

    // updating input to intermediate
    checkCudaErrors(hipMemcpy(d_in, d_tmp, sizeof(int) * grid_size, hipMemcpyDeviceToDevice));

    // Updating grid_size to reflect how many blocks we now want to compute on
    prev_grid_size = grid_size;
    grid_size = num_elems / BLOCK_SIZE + 1;      

    // updating intermediate
    checkCudaErrors(hipFree(d_tmp));
    checkCudaErrors(hipMalloc(&d_tmp, sizeof(int) * grid_size));
  } while(num_elems > BLOCK_SIZE);

  // computing rest
  reduce_kernel<<<1, num_elems>>>(d_out, d_in, prev_grid_size);
}

void par_reduce(unsigned int* const h_out, 
             unsigned int* const h_in,
             const size_t NUM_ELEMS) {
  const int BLOCK_SIZE  = 512;
  const unsigned int ARRAY_BYTES = sizeof(unsigned int) * NUM_ELEMS;

  // device memory
  unsigned int *d_in, *d_out;
  checkCudaErrors(hipMalloc((void **) &d_in, ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_out,   ARRAY_BYTES));

  // Transfer the arrays to the GPU
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
  // run kernel
  reduce_wrapper(d_out, d_in, NUM_ELEMS, BLOCK_SIZE);
  // copy values to host
  hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
  // free device memory
  hipFree(d_in); hipFree(d_out);
}
