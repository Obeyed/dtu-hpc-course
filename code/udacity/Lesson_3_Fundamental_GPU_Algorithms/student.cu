#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/


#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void simple_histo(int *d_bins, const int *d_in, const size_t BIN_COUNT, size_t ARRAY_SIZE, float min_logLum, float range_logLum)
{
  unsigned int myId = threadIdx.x + blockDim.x + blockIdx.x;
  // checking for out-of-bounds
  if (myId>=ARRAY_SIZE)
  {
    return;
  }

  unsigned int myItem = d_in[myId];
  unsigned int myBin = min(static_cast<unsigned int>(BIN_COUNT - 1),
               static_cast<unsigned int>(((d_in[mid]-min_logLum) / range_logLum) * bin_count));
  atomicAdd(&(d_bins[myBin]), 1);
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  // 1)
  const size_t ARRAY_SIZE = numRows * numCols;
  const size_t ARRAY_BYTES = sizeof(float) * ARRAY_SIZE;

  // debugging
  printf("numRows: %d\n", numRows);
  printf("numCols: %d\n", numCols);
  printf("numBins: %d\n", numBins);  
  printf("ARRAY_SIZE: %d\n", ARRAY_SIZE);
  printf("ARRAY_BYTES: %d\n", ARRAY_BYTES);

  min_logLum = reduce_minmax(d_logLuminance, ARRAY_SIZE, 0);
  max_logLum = reduce_minmax(d_logLuminance, ARRAY_SIZE, 1);

  // 2)
  float range_logLum = max_logLum - min_logLum;

  // debugging!
  printf("got min of %f\n", min_logLum);
  printf("got max of %f\n", max_logLum);
  printf("got range of %f\n", range_logLum);

  // 3)
  unsigned int* d_bins;
  const size_t BIN_BYTES = sizeof(size_t)*numBins;
  checkCudaErrors(hipMalloc(&d_bins, BIN_BYTES));    
  checkCudaErrors(hipMemset(d_bins, 0, BIN_BYTES));
  num_threads = 1024;
  simple_histo<<<ARRAY_SIZE/num_threads+1, num_threads>>>(d_bins, d_logLuminance, numBins, ARRAY_SIZE, min_logLum, range_logLum);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  unsigned int * h_out = (unsigned int *)malloc(BIN_BYTES);
  hipMemcpy(&h_out, d_bins, BIN_BYTES, hipMemcpyDeviceToHost);

  // debugging
  for(int i = 0; i < 100; i++)
  {
      printf("hist out %d\n", h_out[i]);
  }

  // Using H&S, so making it "excluding"
  for(int i = numBins-1; i > 0 ; i--)
  {
      h_out[i] = h_out[i-1];
  }
  h_out[0] = 0;
  hipMemcpy(d_bins, &h_out, BIN_BYTES, hipMemcpyHostToDevice);
  // 4)
  

  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */
}