#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

const unsigned int BLOCK_SIZE = 1024;

__global__
void fastHisto_kernel(unsigned int ** d_out,
               unsigned int * d_in,
               unsigned int SIZE){
  unsigned int mid = threadIdx.x + blockIdx.x*blockDim.x;
  if(mid>=SIZE) return;
  unsigned int myVal = d_in[mid];
  atomicAdd(&(d_out[blockIdx.x][myVal]),1);
}

__global__
void fill_pointers_kernel(unsigned int * d_in, unsigned int SIZE, unsigned int rows){
  unsigned int mid = threadIdx.x + blockDim.x * blockIdx.x;
  if(mid>=SIZE) return;
  d_in[mid] = d_in[0] + rows*mid;
}

__global__
void transpose_kernel(unsigned int ** d_out, unsigned int ** d_in, unsigned int GRID_SIZE, unsigned int OUT_SIZE){
  for(int j=0; j < OUT_SIZE; j++)
    for(int i=0; i < GRID_SIZE; i++)
      d_out[j][i] = d_in[i][j]; // out(j,i) = in(i,j)
}

/* -------- KERNEL -------- */
__global__
void reduce_kernel(unsigned int * d_out, unsigned int * d_in, unsigned int SIZE, unsigned int bin, bool last)
{
  // position and threadId
  unsigned int tid = threadIdx.x;
  unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  // do reduction in global memory
  for (unsigned int s = blockDim.x / 2; s>0; s>>=1)
  {
    if ((tid < s) && (mid+s < SIZE)) // Handling out of bounds
        d_in[mid] = d_in[mid] + d_in[mid+s];
    __syncthreads();
  }

  // only thread 0 writes result, as thread
  if ((tid==0) && (mid < SIZE))
  	if(last==false)
      d_out[blockIdx.x] = d_in[mid];
    else
      d_out[bin] = d_in[mid];
}

/* -------- REDUCE KERNEL WRAPPER -------- */
void reduce(unsigned int * d_out, unsigned int * d_in, unsigned int SIZE, unsigned int bin)
{
  // Setting up blocks and intermediate result holder
  unsigned int SIZE_REDUCE = SIZE;
  unsigned int GRID_SIZE_REDUCE = SIZE/BLOCK_SIZE + ((SIZE % BLOCK_SIZE)?1:0);
  unsigned int * d_intermediate;
  hipMalloc(&d_intermediate, sizeof(unsigned int)*GRID_SIZE_REDUCE);
  // Recursively solving, will run approximately log base BLOCK_SIZE times.
  do
  {
    reduce_kernel<<<GRID_SIZE_REDUCE, BLOCK_SIZE>>>(d_intermediate, d_in, SIZE_REDUCE, false, bin);

    // Updating SIZE
    SIZE_REDUCE = GRID_SIZE_REDUCE;//SIZE / NUM_THREADS + SIZE_REST;

    // Updating input to intermediate
    std::swap(d_in, d_intermediate);

    // Updating NUM_BLOCKS to reflect how many blocks we now want to compute on
    GRID_SIZE_REDUCE = SIZE_REDUCE/BLOCK_SIZE + ((SIZE_REDUCE % BLOCK_SIZE)?1:0);
  }
  while(SIZE_REDUCE > BLOCK_SIZE); // if it is too small, compute rest.

  // Computing rest
  reduce_kernel<<<1, SIZE>>>(d_out, d_in, SIZE_REDUCE, true, bin);
  hipFree(d_intermediate);
}

void merge(unsigned int * d_out, unsigned int * d_in, unsigned int SIZE, unsigned int bin){
  reduce(d_out, d_in, SIZE, bin);
}

void fastHisto(unsigned int * d_out, unsigned int * d_in, unsigned int IN_SIZE, unsigned int GRID_SIZE, unsigned int OUT_SIZE){
  //Setting up major histo
  unsigned int ** d_out_all, ** d_out_all_trans;
  unsigned int GRID_SIZE_ALL = GRID_SIZE * OUT_SIZE;
  unsigned int GRID_BYTES_ALL = GRID_SIZE_ALL * sizeof(unsigned int);
  hipMalloc(&d_out_all, GRID_BYTES_ALL);
  hipMemset(d_out_all, 0, GRID_BYTES_ALL);
  fastHisto_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_out_all, d_in, IN_SIZE);
  hipMalloc(&d_out_all_trans, GRID_BYTES_ALL);
  transpose_kernel<<<1, 1>>>(d_out_all_trans, d_out_all, GRID_SIZE, OUT_SIZE);
  hipFree(d_out_all);

  // Merging histograms reduce
  for(unsigned int bin = 0; bin<OUT_SIZE; bin++){
    merge(d_out, d_out_all_trans[bin], GRID_SIZE, bin);
  }
  hipFree(d_out_all_trans);
}

int main(int argc, char **argv){
  printf("---STARTED---\n");
  // Vars
  unsigned int IN_SIZE;
  unsigned int IN_BYTES;
  unsigned int OUT_SIZE;
  unsigned int OUT_BYTES;
  unsigned int GRID_SIZE;
  unsigned int h_filler;
  unsigned int sum;

  for(unsigned int rounds = 2; rounds<8; rounds++){
    IN_SIZE = 1<<8;
    IN_BYTES = sizeof(unsigned int) * IN_SIZE;
    OUT_SIZE = 1<<rounds;
    OUT_BYTES = sizeof(unsigned int) * OUT_SIZE;
    GRID_SIZE = IN_SIZE/BLOCK_SIZE + ((IN_SIZE % BLOCK_SIZE)?1:0);

    // Generate the input array on host
    unsigned int * h_in = (unsigned int *)malloc(IN_BYTES);
    unsigned int * h_out = (unsigned int *)malloc(OUT_BYTES);
    for (h_filler = 0; h_filler<IN_SIZE; h_filler++) {h_in[h_filler] = h_filler;}

    // Declare GPU memory pointers
    printf("\n@@@ROUND@@@: %d\n", rounds);
    printf("---IN_SIZE---: %d\n", IN_SIZE);
    printf("---IN_BYTES---: %d\n", IN_BYTES);
    printf("---OUT_SIZE---: %d\n", OUT_SIZE);
    printf("---OUT_BYTES---: %d\n", OUT_BYTES);
    printf("---BLOCK_SIZE---: %d\n", BLOCK_SIZE);
    printf("---GRID_SIZE---: %d\n", GRID_SIZE);

    unsigned * d_in;
	unsigned * d_out;
	// Allocate GPU memory
    hipMalloc(&d_in, IN_BYTES);
    printf("---ALLOCATED D_IN---\n");
    hipMalloc(&d_out, OUT_BYTES);
    printf("---ALLOCATED D_OUT---\n");

    // Transfer the arrays to the GPU
    hipMemcpy(d_in, h_in, IN_BYTES, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // running the code on the GPU
    hipMemset(d_out, 0, OUT_BYTES);
    fastHisto(d_out, d_in, IN_SIZE, GRID_SIZE, OUT_SIZE);
//    simple_histo<<<GRID_SIZE, BLOCK_SIZE>>>(d_out, d_in, OUT_SIZE, IN_SIZE);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // calculating time
    float elapsedTime = .0f;
    hipEventElapsedTime(&elapsedTime, start, stop);
    //elapsedTime = elapsedTime / ((float) times);
    printf(" time: %.5f\n", elapsedTime);

    // Copy back to HOST
    hipMemcpy(h_out, d_out, OUT_BYTES, hipMemcpyDeviceToHost);
    sum = 0;
    for(unsigned int i = 0; i<OUT_SIZE; i++) sum += h_out[i];
    for(unsigned int i = 0; (i<OUT_SIZE) && (i<10); i++){
      printf("bin %d: count %d\n", i, h_out[i]);
    }
    printf("%d\n", sum);

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);
  }
  return 0;
}