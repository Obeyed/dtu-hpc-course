#include "hip/hip_runtime.h"

#include <stdio.h>      /* printf, scanf, puts, NULL */
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <string.h>
#include "utils.h"
#include "radix_sort.h"

// CONSTANTS
const unsigned int NUM_ELEMS    = 1 << 10;
const unsigned int NUM_BINS     = 100;
const unsigned int ARRAY_BYTES  = sizeof(unsigned int) * NUM_ELEMS;
const unsigned int TOTAL_BIN_BYTES  = sizeof(unsigned int) * NUM_BINS;

const dim3 BLOCK_SIZE(1 << 8);
const dim3 GRID_SIZE(NUM_ELEMS / BLOCK_SIZE.x + 1);

const unsigned int COARSER_SIZE = NUM_BINS / 10;
const unsigned int COARSER_BYTES = sizeof(unsigned int) * COARSER_SIZE;
const unsigned int MAX_NUMS = 1000;

__global__
void fire_up_local_bins(unsigned int* const d_out,
                        const unsigned int* const d_bins,
                        const unsigned int l_start,
                        const int l_end) {
  if (l_end < 0) return; // means that no values are in coarsed bin

  const unsigned int l_pos = l_start + threadIdx.x + blockIdx.x * blockDim.x;
  if (l_pos < l_start || l_pos >= l_end) return;

  const unsigned int bin = d_bins[l_pos];
  // read some into shared memory
  // atomic adds
  // write to global memory
  atomicAdd(&(d_out[bin]), 1);
}

__global__
void compute_coarse_bin_mapping(const unsigned int* const d_in,
                                unsigned int* const d_out,
                                const size_t COARSE) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= NUM_ELEMS) return;

  d_out[mid] = d_in[mid] / COARSE;
}

__global__
void compute_bin_mapping(const unsigned int* const d_in,
                         unsigned int* const d_out) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= NUM_ELEMS) return;

  d_out[mid] = d_in[mid] % NUM_BINS;
}

__global__
void find_positions_mapping_kernel(unsigned int* const d_out,
                                   const unsigned int* const d_in) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if ((mid >= NUM_ELEMS) || (mid == 0)) return;

  if (d_in[mid] != d_in[mid-1])
    d_out[d_in[mid]] = mid;
}

void init_rand(unsigned int* const h_in) {
  /* initialize random seed: */
  srand(time(NULL));

  /* generate values between 0 and 999: */
  for (int i = 0; i < NUM_ELEMS; i++)
    h_in[i] = rand() % MAX_NUMS;
}

void print(const unsigned int* const h_in,
           const unsigned int* const h_bins,
           const unsigned int* const h_coarse_bins,
           const unsigned int* const h_positions) {
  const unsigned int WIDTH = 4;

  for(int i = 0; i < WIDTH; i++)
    printf("input\tbin\tcoarse\t\t");
  printf("\n");

  for (int i = 0; i < NUM_ELEMS; i++)
    printf("%u\t%u\t%u%s", 
        h_in[i], 
        h_bins[i], 
        h_coarse_bins[i], 
        ((i % WIDTH == (WIDTH-1)) ? "\n" : "\t\t"));
  printf("\n");

  printf("positions:\n");
  for (int i = 0; i < COARSER_SIZE; i++)
    printf("%u\t", h_positions[i]);
  printf("\n");
}

void sort(unsigned int*& h_coarse_bins, 
          unsigned int*& h_bins, 
          unsigned int*& h_values) {
  const unsigned int NUM_ARRAYS = 3;
  // set up pointers
  unsigned int** to_be_sorted = new unsigned int*[NUM_ARRAYS];
  to_be_sorted[0] = h_coarse_bins;
  to_be_sorted[1] = h_bins;
  to_be_sorted[2] = h_values;

  unsigned int** sorted = radix_sort(to_be_sorted, NUM_ARRAYS, NUM_ELEMS);

  // update pointers
  h_coarse_bins = sorted[0];
  h_bins = sorted[1];
  h_values = sorted[2];
}

int main(int argc, char **argv) {
  printf("## STARTING ##\n");
  printf("blocks: %u\tthreads: %u\t COARSER_SIZE: %u", GRID_SIZE.x, BLOCK_SIZE.x, COARSER_SIZE);
  printf("\n\n");

  // create random values
  unsigned int* h_values = new unsigned int[NUM_ELEMS];
  init_rand(h_values);
  // host memory
  unsigned int* h_bins = new unsigned int[NUM_ELEMS];
  unsigned int* h_coarse_bins = new unsigned int[NUM_ELEMS];
  unsigned int* h_histogram = new unsigned int[NUM_BINS];
  unsigned int* h_positions = new unsigned int[COARSER_SIZE];

  //copy values to device memory
  unsigned int* d_values, * d_bins, * d_coarse_bins, * d_positions;
  checkCudaErrors(hipMalloc((void **) &d_values, ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_bins,   ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_coarse_bins, ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_positions, ARRAY_BYTES));

  // copy host memory to device
  checkCudaErrors(hipMemcpy(d_values, h_values,  ARRAY_BYTES, hipMemcpyHostToDevice));

  // compute bin id
  compute_bin_mapping<<<GRID_SIZE, BLOCK_SIZE>>>(d_values, d_bins);
  // move memory to host
  checkCudaErrors(hipMemcpy(h_bins, d_bins, ARRAY_BYTES, hipMemcpyDeviceToHost));

  // compute coarse bin id
  compute_coarse_bin_mapping<<<GRID_SIZE, BLOCK_SIZE>>>(d_bins, d_coarse_bins, COARSER_SIZE);
  // move memory to host
  checkCudaErrors(hipMemcpy(h_coarse_bins, d_coarse_bins, ARRAY_BYTES, hipMemcpyDeviceToHost));

  // sort
  sort(h_coarse_bins, h_bins, h_values);
  checkCudaErrors(hipMemcpy(d_coarse_bins, h_coarse_bins,  ARRAY_BYTES, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_bins,        h_bins,         ARRAY_BYTES, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_values,      h_values,       ARRAY_BYTES, hipMemcpyHostToDevice));

  // find starting position for each coarsed bin
  hipMemset(d_positions, 0, COARSER_BYTES);
  checkCudaErrors(hipMemcpy(h_positions, d_positions, COARSER_BYTES, hipMemcpyDeviceToHost));

  // find positions of separators
  find_positions_mapping_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_positions, d_coarse_bins);
  checkCudaErrors(hipMemcpy(h_positions, d_positions, COARSER_BYTES, hipMemcpyDeviceToHost));
  
  print(h_values, h_bins, h_coarse_bins, h_positions);

  // ####
  unsigned int* d_bin_grid;
  // created entire bin grid in first run
  // only access relevant elements in kernel
  // based on bin_size and bin_start
  checkCudaErrors(hipMalloc((void **) &d_bin_grid, TOTAL_BIN_BYTES));
  checkCudaErrors(hipMemset(d_bin_grid, 0, TOTAL_BIN_BYTES));

  // make some local bins
  unsigned int local_bin_start = 0;
  unsigned int local_bin_end = h_positions[1];
  int amount = local_bin_end - local_bin_start;
  // calculate local grid size
  unsigned int grid_size = local_bin_end / BLOCK_SIZE.x + 1;

  fire_up_local_bins<<<grid_size, BLOCK_SIZE>>>(d_bin_grid, d_bins, local_bin_start, local_bin_end);

  for (unsigned int i = 1; i < COARSER_SIZE - 1; i++) {
    // make some local bins
    local_bin_start = h_positions[i];
    local_bin_end   = h_positions[i+1];
    amount = local_bin_end - local_bin_start;
    // calculate local grid size
    grid_size = local_bin_end / BLOCK_SIZE.x + 1;

    if (amount > 0)
      fire_up_local_bins<<<grid_size, BLOCK_SIZE>>>(d_bin_grid, d_bins, local_bin_start, local_bin_end);
  }


  // do final loop
  local_bin_start = h_positions[COARSER_SIZE-1];
  local_bin_end   = NUM_ELEMS;
  amount = local_bin_end - local_bin_start;
  // calculate local grid size
  grid_size = local_bin_end / BLOCK_SIZE.x + 1;

  if (amount > 0)
    fire_up_local_bins<<<grid_size, BLOCK_SIZE>>>(d_bin_grid, d_bins, local_bin_start, local_bin_end);

  checkCudaErrors(hipMemcpy(h_histogram, d_bin_grid, TOTAL_BIN_BYTES, hipMemcpyDeviceToHost));

  printf("\n");
  for (int j = 0; j < NUM_BINS; j++)
    printf("%d:%u\t%s", 
        j,
        h_histogram[j], 
        ((j % 6 == 5) ? "\n" : "\t\t"));
  printf("\n");

  //#####

  hipFree(d_bin_grid); hipFree(d_values); hipFree(d_positions);
  hipFree(d_coarse_bins); hipFree(d_bins);

  printf("## DONE ##\n");

  return 0;
}

