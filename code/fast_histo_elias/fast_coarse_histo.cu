#include "hip/hip_runtime.h"

#include <stdio.h>      /* printf, scanf, puts, NULL */
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <string.h>
#include "utils.h"

// CONSTANTS
const unsigned int NUM_ELEMS    = 1 << 10;
const unsigned int NUM_BINS     = 100;
const unsigned int ARRAY_BYTES  = sizeof(unsigned int) * NUM_ELEMS;

const unsigned int COARSER = 10;

const dim3 BLOCK_SIZE(1 << 8);
const dim3 GRID_SIZE(NUM_ELEMS / BLOCK_SIZE.x);
const dim3 GRID_SIZE_COARSED(GRID_SIZE.x / COARSER);

__global__
void compute_coarse_bin_mapping(const unsigned int* const d_in,
                                unsigned int* const d_out,
                                const size_t BLOCKS) {
  unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= NUM_ELEMS) return;

  d_out[mid] = d_in[mid] % BLOCKS;
}

__global__
void compute_bin_mapping(const unsigned int* const d_in,
                         unsigned int* const d_out) {
  unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= NUM_ELEMS) return;

  d_out[mid] = d_in[mid] % NUM_BINS;
}

void init_rand(unsigned int* const h_in) {
  /* initialize random seed: */
  srand(time(NULL));

  /* generate values between 0 and 999: */
  for (int i = 0; i < NUM_ELEMS; i++)
    h_in[i] = rand() % 1000;
}

void print(const unsigned int* const h_in,
           const unsigned int* const h_bins,
           const unsigned int* const h_coarse_bins) {
  const unsigned int WIDTH = 6;

  for(int i = 0; i < WIDTH; i++)
    printf("input\tbin\tcoarse\t\t");
  printf("\n");

  for (int i = 0; i < NUM_ELEMS; i++)
    printf("%u\t%u\t%u%s", 
        h_in[i], 
        h_bins[i], 
        h_coarse_bins[i], 
        ((i % WIDTH == (WIDTH-1)) ? "\n" : "\t\t"));
  printf("\n");
  printf("SHOULD RETURN NOW");
}

int main(int argc, char **argv) {
  printf("## STARTING ##\n");
  printf("blocks: %u\tthreads: %u\t coarsed blocks: %u", GRID_SIZE.x, BLOCK_SIZE.x, GRID_SIZE_COARSED.x);

  printf("\n\n");

  // create random values
  unsigned int* h_values = new unsigned int[NUM_ELEMS];
  init_rand(h_values);
  // host memory
  unsigned int* h_bins = new unsigned int[NUM_ELEMS];
  unsigned int* h_coarse_bins = new unsigned int[NUM_ELEMS];

  //copy values to device memory
  unsigned int* d_values, 
              * d_bins,
              * d_coarse_bins;
  checkCudaErrors(hipMalloc((void **) &d_values, ARRAY_BYTES));
  checkCudaErrors(hipMemcpy(d_values, h_values,  ARRAY_BYTES, hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void **) &d_bins,   ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_coarse_bins,   ARRAY_BYTES));

  // compute bin id
  compute_bin_mapping<<<GRID_SIZE, BLOCK_SIZE>>>(d_values, d_bins);
  // move memory to host
  checkCudaErrors(hipMemcpy(h_bins, d_bins, ARRAY_BYTES, hipMemcpyDeviceToHost));

  // compute coarse bin id
  compute_coarse_bin_mapping<<<GRID_SIZE, BLOCK_SIZE>>>(d_bins, d_coarse_bins, GRID_SIZE.x);
  // move memory to host
  checkCudaErrors(hipMemcpy(h_coarse_bins, d_coarse_bins, ARRAY_BYTES, hipMemcpyDeviceToHost));

  // sort

  // send coarse bin to each block
  // atomicAdd bins in shared memory
  
  // combine bins and write to global memory


  print(h_values, h_bins, h_coarse_bins);

  printf("## DONE ##");

  return 0;
}
