#include "hip/hip_runtime.h"

#include <stdio.h>      /* printf, scanf, puts, NULL */
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <string.h>
#include "utils.h"
#include "radix_sort.h"

// CONSTANTS
const unsigned int NUM_ELEMS    = 1 << 5;
const unsigned int NUM_BINS     = 100;
const unsigned int ARRAY_BYTES  = sizeof(unsigned int) * NUM_ELEMS;

const dim3 BLOCK_SIZE(1 << 8);
const dim3 GRID_SIZE(NUM_ELEMS / BLOCK_SIZE.x + 1);

const unsigned int COARSER = NUM_BINS / 10;
const unsigned int MAX_NUMS = 1000;

__global__
void compute_coarse_bin_mapping(const unsigned int* const d_in,
                                unsigned int* const d_out,
                                const size_t COARSE) {
  unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= NUM_ELEMS) return;

  d_out[mid] = d_in[mid] / COARSE;
}

__global__
void compute_bin_mapping(const unsigned int* const d_in,
                         unsigned int* const d_out) {
  unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= NUM_ELEMS) return;

  d_out[mid] = d_in[mid] % NUM_BINS;
}

void init_rand(unsigned int* const h_in) {
  /* initialize random seed: */
  srand(time(NULL));

  /* generate values between 0 and 999: */
  for (int i = 0; i < NUM_ELEMS; i++)
    h_in[i] = rand() % MAX_NUMS;
}

void print(const unsigned int* const h_in,
           const unsigned int* const h_bins,
           const unsigned int* const h_coarse_bins) {
  const unsigned int WIDTH = 6;

  for(int i = 0; i < WIDTH; i++)
    printf("input\tbin\tcoarse\t\t");
  printf("\n");

  for (int i = 0; i < NUM_ELEMS; i++)
    printf("%u\t%u\t%u%s", 
        h_in[i], 
        h_bins[i], 
        h_coarse_bins[i], 
        ((i % WIDTH == (WIDTH-1)) ? "\n" : "\t\t"));
  printf("\n");
}

int main(int argc, char **argv) {
  printf("## STARTING ##\n");
  printf("blocks: %u\tthreads: %u\t coarser: %u", GRID_SIZE.x, BLOCK_SIZE.x, COARSER);

  printf("\n\n");

  // create random values
  unsigned int* h_values = new unsigned int[NUM_ELEMS];
  init_rand(h_values);
  // host memory
  unsigned int* h_bins = new unsigned int[NUM_ELEMS];
  unsigned int* h_coarse_bins = new unsigned int[NUM_ELEMS];

  //copy values to device memory
  unsigned int* d_values, 
              * d_bins,
              * d_coarse_bins;
  checkCudaErrors(hipMalloc((void **) &d_values, ARRAY_BYTES));
  checkCudaErrors(hipMemcpy(d_values, h_values,  ARRAY_BYTES, hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void **) &d_bins,   ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_coarse_bins,   ARRAY_BYTES));

  // compute bin id
  compute_bin_mapping<<<GRID_SIZE, BLOCK_SIZE>>>(d_values, d_bins);
  // move memory to host
  checkCudaErrors(hipMemcpy(h_bins, d_bins, ARRAY_BYTES, hipMemcpyDeviceToHost));

  // compute coarse bin id
  compute_coarse_bin_mapping<<<GRID_SIZE, BLOCK_SIZE>>>(d_bins, d_coarse_bins, COARSER);
  // move memory to host
  checkCudaErrors(hipMemcpy(h_coarse_bins, d_coarse_bins, ARRAY_BYTES, hipMemcpyDeviceToHost));

  printf("BEFORE SORTING:\n");
  print(h_values, h_bins, h_coarse_bins);

  // sort
  const unsigned int NUM_ARRAYS = 3;
  unsigned int** all_arrays = new unsigned int*[NUM_ARRAYS];
  all_arrays[0] = h_coarse_bins;
  all_arrays[1] = h_bins;
  all_arrays[2] = h_values;

  unsigned int** sorted = radix_sort(all_arrays, NUM_ARRAYS, NUM_ELEMS);
  printf("FROM HISTO:\n");
  printf("sorted[0] = %u\n", sorted[0]);
  printf("sorted[1] = %u\n", sorted[1]);
  printf("sorted[2] = %u\n", sorted[2]);

  h_coarse_bins = sorted[0];
  h_bins = sorted[1];
  h_values = sorted[2];

  printf("updated[0] = %u\n", h_coarse_bins);
  printf("updated[1] = %u\n", h_bins);
  printf("updated[2] = %u\n", h_values);


  // send coarse bin to each block
  // atomicAdd bins in shared memory
  
  // combine bins and write to global memory


  printf("AFTER SORTING:\n");
  print(h_values, h_bins, h_coarse_bins);

  printf("## DONE ##\n");

  return 0;
}
