#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>


// Computes the sum of elements in d_in in shared memory
__global__ 
void shared_reduce_kernel(unsigned int* const d_out,
                          unsigned int* const d_in,
                          const size_t NUM_ELEMS) {
  unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;

  if (pos >= NUM_ELEMS) return;

  extern __shared__ unsigned int sdata[]; // allocate shared memory
  sdata[tid] = d_in[pos];                 // each thread loads global to shared
  __syncthreads();                        // make sure all threads are done

  for (unsigned int s = blockDim.x / 2; s > 0; s >>=1) {
    if ((tid < s) && ((pos + s) < NUM_ELEMS))
      sdata[tid] +=  sdata[tid + s];      // perform operations on shared memory
    __syncthreads();
  }

  // only thread 0 writes result, as thread
  if ((tid == 0) && (pos < NUM_ELEMS))
    d_out[blockIdx.x] = sdata[0];         // copy shared back to global
}

/* -------- KERNEL -------- */
__global__ void reduce_kernel(unsigned int * d_out, unsigned int * d_in, unsigned int SIZE)
{
  // position and threadId
  unsigned int tid = threadIdx.x;
  unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;

  // do reduction in global memory
  for (unsigned int s = blockDim.x / 2; s>0; s>>=1)
  {
    if ((tid < s) && (mid+s < SIZE)) // Handling out of bounds
        d_in[mid] = d_in[mid] + d_in[mid+s];
    __syncthreads();
  }

  // only thread 0 writes result, as thread
  if ((tid==0) && (mid < SIZE))
    d_out[blockIdx.x] = d_in[mid];
}

/* -------- KERNEL WRAPPER -------- */
void reduce(unsigned int * d_out, unsigned int * d_in, unsigned int SIZE, unsigned int NUM_THREADS)
{
  // Setting up blocks and intermediate result holder
  unsigned int NUM_BLOCKS = SIZE/NUM_THREADS + ((SIZE % NUM_THREADS)?1:0);
  unsigned int * d_intermediate_in;
  unsigned int * d_intermediate_out;
  hipMalloc(&d_intermediate_in, sizeof(unsigned int)*SIZE);
  hipMalloc(&d_intermediate_out, sizeof(unsigned int)*NUM_BLOCKS);
  hipMemcpy(d_intermediate_in, d_in, sizeof(unsigned int)*SIZE, hipMemcpyDeviceToDevice);

  // calculate shared memory
  const unsigned int SMEM = NUM_THREADS * sizeof(unsigned int);
  // Recursively solving, will run approximately log base NUM_THREADS times.
  do
  {
    shared_reduce_kernel<<<NUM_BLOCKS, NUM_THREADS, (NUM_THREADS * sizeof(unsigned int))>>>(d_intermediate_out, d_intermediate_in, SIZE);

    // Updating SIZE
    SIZE = NUM_BLOCKS;//SIZE / NUM_THREADS + SIZE_REST;

    // Updating input to intermediate
    hipMemcpy(d_intermediate_in, d_intermediate_out, sizeof(unsigned int)*NUM_BLOCKS, hipMemcpyDeviceToDevice);

    // Updating NUM_BLOCKS to reflect how many blocks we now want to compute on
    NUM_BLOCKS = SIZE/NUM_THREADS + ((SIZE % NUM_THREADS)?1:0);

  }
  while(SIZE > NUM_THREADS); // if it is too small, compute rest.

  // Computing rest
  shared_reduce_kernel<<<1, SIZE, SMEM>>>(d_out, d_intermediate_out, SIZE);
  hipFree(d_intermediate_in);
  hipFree(d_intermediate_out);
}

/* -------- MAIN -------- */
int main(int argc, char **argv)
{
  std::ofstream myfile;
  myfile.open ("par_reduce.csv");
  // Setting NUM_THREADS
  const unsigned int times = 10;
  for (unsigned int rounds = 0; rounds<30; rounds++)
  {
//    printf("Round: %d\n", rounds);
    unsigned int NUM_THREADS = 1<<10;
    // Making non-bogus data and setting it on the GPU
    unsigned int SIZE = 1<<rounds;
    unsigned int * d_in;
    unsigned int * d_out;
    hipMalloc(&d_in, sizeof(unsigned int)*SIZE);
    hipMalloc(&d_out, sizeof(unsigned int)*SIZE);
    unsigned int * h_in = (unsigned int *)malloc(SIZE*sizeof(int));
    for (unsigned int i = 0; i <  SIZE; i++) h_in[i] = 1;
    hipMemcpy(d_in, h_in, sizeof(unsigned int)*SIZE, hipMemcpyHostToDevice);

    // Running kernel wrapper
    // setting up time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // kernel time!!!
    hipEventRecord(start, 0);

    for (unsigned int i = 0; i < times; i++)
    {
      reduce(d_out, d_in, SIZE, NUM_THREADS);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // calculating time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);    
    elapsedTime = elapsedTime / ((float) times);
//    printf("time!: %.5f\n", elapsedTime);
    unsigned int h_out;
    hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);
//    printf("%d \n", h_out);
    myfile << elapsedTime << "," << std::endl;
  }
  myfile.close();
  return 0;
}
