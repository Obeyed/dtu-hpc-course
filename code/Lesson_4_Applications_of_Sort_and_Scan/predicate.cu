#include "hip/hip_runtime.h"
// Create predicate array for HW4

#include "utils.h"
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>

/*
 * Calculate if LSB is 0.
 * 1 if true, 0 otherwise.
 */
__global__
void predicate_kernel(unsigned int *d_predicate,
                      unsigned int *d_val_src,
                      const size_t numElems) {
  unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;
  d_predicate[mid] = (int)((d_val_src[mid] & 1) == 0);
}

int main(void) {
  size_t numElems = 16;
  int ARRAY_BYTES = sizeof(unsigned int) * numElems;

  // device memory
  unsigned int* d_val_src;
  unsigned int* d_predicate;
  checkCudaErrors(hipMalloc((void **) &d_val_src,   ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_predicate, ARRAY_BYTES));

  // input array
  unsigned int* h_input = new unsigned int[numElems];
  h_input[0]  = 39; h_input[1]  = 21; h_input[2]  = 84; h_input[3]  = 40;
  h_input[4]  = 78; h_input[5]  = 85; h_input[6]  = 13; h_input[7]  = 4;
  h_input[8]  = 47; h_input[9]  = 45; h_input[10] = 91; h_input[11] = 60;
  h_input[12] = 74; h_input[13] = 8;  h_input[14] = 44; h_input[15] = 53;
  checkCudaErrors(hipMemcpy(d_val_src, h_input, ARRAY_BYTES, hipMemcpyHostToDevice));

  // kernel call
  predicate_kernel<<<2,8>>>(d_predicate, d_val_src, numElems);

  // debugging
  unsigned int *h_predicate = new unsigned int[numElems];
  checkCudaErrors(hipMemcpy(h_predicate, d_predicate,  ARRAY_BYTES, hipMemcpyDeviceToHost));

  printf("INPUT \t PRED \n");
  for (int i = 0; i < numElems; i++)
    printf("%u \t %u\n", h_input[i], h_predicate[i]);

  return 0;
}
