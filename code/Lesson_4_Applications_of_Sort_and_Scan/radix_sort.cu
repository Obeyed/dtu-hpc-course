#include "hip/hip_runtime.h"
// Create predicate array for HW4
#include "radix_sort.h"

/*
 * Calculate if LSB is 0.
 * 1 if true, 0 otherwise.
 */
__global__
void predicate_kernel(unsigned int* const d_predicate,
                      const unsigned int* const d_val_src,
                      const size_t numElems,
                      const unsigned int i) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  d_predicate[mid] = (int)(((d_val_src[mid] & (1 << i)) >> i) == 0);
}

__global__
void inclusive_sum_scan_kernel(unsigned int* const d_out,
                               const unsigned int* const d_in,
                               const int step,
                               const size_t numElems) {
  const int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

	int toAdd = (((mid - step) < 0) ? 0 : d_in[mid - step]);
  d_out[mid] = d_in[mid] + toAdd;
}

__global__
void right_shift_array_kernel(unsigned int* const d_out,
                       const unsigned int* const d_in,
                       const size_t numElems) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  d_out[mid] = (mid == 0) ? 0 : d_in[mid - 1];
}

__global__
void toggle_predicate_kernel(unsigned int* const d_out, 
                             const unsigned int* const d_predicate,
                             const size_t numElems) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  d_out[mid] = ((d_predicate[mid]) ? 0 : 1);
}

__global__
void add_splitter_map_kernel(unsigned int* const d_out,
                             const unsigned int* const shift, 
                             const size_t numElems) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  d_out[mid] += shift[0];
}

__global__ 
void reduce_kernel(unsigned int* const d_out,
                   unsigned int* const d_in,
                   const size_t numElems) {
  unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;

  for (unsigned int s = blockDim.x / 2; s > 0; s >>=1) {
    if ((tid < s) && ((pos + s) < numElems))
      d_in[pos] = d_in[pos] + d_in[pos + s];
    __syncthreads();
  }

  // only thread 0 writes result, as thread
  if ((tid == 0) && (pos < numElems))
    d_out[blockIdx.x] = d_in[pos];
}

__global__
void map_kernel(unsigned int* const d_out,
                const unsigned int* const d_in,
                const unsigned int* const d_predicate,
                const unsigned int* const d_sum_scan_0,
                const unsigned int* const d_sum_scan_1,
                const size_t numElems) {
  const unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  const unsigned int pos = ((d_predicate[mid]) ? d_sum_scan_0[mid] : d_sum_scan_1[mid]);
  d_out[pos] = d_in[mid];
}

void reduce_wrapper(unsigned int* const d_out,
                    unsigned int* const d_in,
                    size_t numElems,
                    int block_size) {
  unsigned int grid_size = numElems / block_size + 1;

  unsigned int* d_tmp;
  checkCudaErrors(hipMalloc(&d_tmp, sizeof(unsigned int) * grid_size));
  checkCudaErrors(hipMemset(d_tmp, 0, sizeof(unsigned int) * grid_size));

  unsigned int prev_grid_size;
  unsigned int remainder = 0;
  // recursively solving, will run approximately log base block_size times.
  do {
    reduce_kernel<<<grid_size, block_size>>>(d_tmp, d_in, numElems);

    remainder = numElems % block_size;
    numElems  = numElems / block_size + remainder;

    // updating input to intermediate
    checkCudaErrors(hipMemcpy(d_in, d_tmp, sizeof(int) * grid_size, hipMemcpyDeviceToDevice));

    // Updating grid_size to reflect how many blocks we now want to compute on
    prev_grid_size = grid_size;
    grid_size = numElems / block_size + 1;      

    // updating intermediate
    checkCudaErrors(hipFree(d_tmp));
    checkCudaErrors(hipMalloc(&d_tmp, sizeof(int) * grid_size));
  } while(numElems > block_size);

  // computing rest
  reduce_kernel<<<1, numElems>>>(d_out, d_in, prev_grid_size);
}

void exclusive_sum_scan(unsigned int* const d_out,
                        const unsigned int* const d_predicate,
                        unsigned int* const d_predicate_tmp,
                        unsigned int* const d_sum_scan,
                        const unsigned int ARRAY_BYTES,
                        const size_t numElems,
                        const int GRID_SIZE,
                        const int BLOCK_SIZE) {
  // copy predicate values to new array
  checkCudaErrors(hipMemcpy(d_predicate_tmp, d_predicate, ARRAY_BYTES, hipMemcpyDeviceToDevice));
  // set all elements to zero 
  checkCudaErrors(hipMemset(d_sum_scan, 0, ARRAY_BYTES));

  // sum scan call
  for (unsigned int step = 1; step < numElems; step *= 2) {
    inclusive_sum_scan_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_sum_scan, d_predicate_tmp, step, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpy(d_predicate_tmp, d_sum_scan, ARRAY_BYTES, hipMemcpyDeviceToDevice));
  }

  // shift to get exclusive scan
  checkCudaErrors(hipMemcpy(d_out, d_sum_scan, ARRAY_BYTES, hipMemcpyDeviceToDevice));
  right_shift_array_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_out, d_sum_scan, numElems);
}

unsigned int* radix_sort(unsigned int* h_input,
                         const size_t numElems) {
  const int BLOCK_SIZE  = 512;
  const int GRID_SIZE   = numElems / BLOCK_SIZE + 1;
  const unsigned int ARRAY_BYTES = sizeof(unsigned int) * numElems;
  const unsigned int BITS_PER_BYTE = 8;

  // host memory
  unsigned int* const h_output = new unsigned int[numElems];

  // device memory
  unsigned int *d_val_src, *d_predicate, *d_sum_scan, *d_predicate_tmp, *d_sum_scan_0, *d_sum_scan_1, *d_predicate_toggle, *d_reduce, *d_map;
  checkCudaErrors(hipMalloc((void **) &d_val_src,          ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_map,              ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_predicate,        ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_predicate_tmp,    ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_predicate_toggle, ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_sum_scan,         ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_sum_scan_0,       ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_sum_scan_1,       ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_reduce, sizeof(unsigned int)));

  // copy host array to device
  checkCudaErrors(hipMemcpy(d_val_src, h_input, ARRAY_BYTES, hipMemcpyHostToDevice));

  for (unsigned int i = 0; i < (BITS_PER_BYTE * sizeof(unsigned int)); i++) {
    // predicate is that LSB is 0
    predicate_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_predicate, d_val_src, numElems, i);

    // calculate scatter addresses from predicates
    exclusive_sum_scan(d_sum_scan_0, d_predicate, d_predicate_tmp, d_sum_scan, ARRAY_BYTES, numElems, GRID_SIZE, BLOCK_SIZE);

    // copy contents of predicate, so we do not change its content
    checkCudaErrors(hipMemcpy(d_predicate_tmp, d_predicate, ARRAY_BYTES, hipMemcpyDeviceToDevice));

    // calculate how many elements had predicate equal to 1
    reduce_wrapper(d_reduce, d_predicate_tmp, numElems, BLOCK_SIZE);

    // toggle predicate values, so we can compute scatter addresses for toggled predicates
    toggle_predicate_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_predicate_toggle, d_predicate, numElems);
    // so we now have addresses for elements where LSB is equal to 1
    exclusive_sum_scan(d_sum_scan_1, d_predicate_toggle, d_predicate_tmp, d_sum_scan, ARRAY_BYTES, numElems, GRID_SIZE, BLOCK_SIZE);
    // shift scatter addresses according to amount of elements that had LSB equal to 0
    add_splitter_map_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_sum_scan_1, d_reduce, numElems);

    // move elements accordingly
    map_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_map, d_val_src, d_predicate, d_sum_scan_0, d_sum_scan_1, numElems);

    // swap pointers, instead of moving elements
    std::swap(d_val_src, d_map);
  }

  // debugging
  checkCudaErrors(hipMemcpy(h_output, d_val_src, ARRAY_BYTES, hipMemcpyDeviceToHost));

  for (int i = 0; i < numElems; i++)
    printf("%u%s", h_output[i], ((i % 8 == 7) ? "\n" : "\t"));

  return h_output;
}
