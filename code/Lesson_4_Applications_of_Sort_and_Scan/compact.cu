#include "hip/hip_runtime.h"
// Create predicate array for HW4

#include "utils.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

/*
 * Calculate if LSB is 0.
 * 1 if true, 0 otherwise.
 */
__global__
void predicate_kernel(unsigned int *d_predicate,
                      unsigned int *d_val_src,
                      const size_t numElems,
                      unsigned int i) {
  unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  d_predicate[mid] = (int)(((d_val_src[mid] & (1 << i)) >> i) == 0);
}

__global__
void inclusive_sum_scan_kernel(unsigned int* d_out,
                               unsigned int* d_in,
                               int step,
                               const size_t numElems) {
  int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

	int toAdd = (((mid - step) < 0) ? 0 : d_in[mid - step]);
  d_out[mid] = d_in[mid] + toAdd;
}

__global__
void right_shift_array(unsigned int* d_out,
                       unsigned int* d_in,
                       size_t numElems) {
  int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  d_out[mid] = (mid == 0) ? 0 : d_in[mid - 1];
}

void DEBUG(unsigned int *device_array, unsigned int ARRAY_BYTES, size_t numElems) {
  unsigned int *h_test  = new unsigned int[numElems];
  checkCudaErrors(hipMemcpy(h_test, device_array, ARRAY_BYTES, hipMemcpyDeviceToHost));

  for (int i = 0; i < numElems; i++)
    printf("%u ", h_test[i]);
  printf("\n");
}

__global__ 
void reduce_kernel(unsigned int * d_out, unsigned int * d_in, int size) {
  unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;

  for (unsigned int s = blockDim.x / 2; s>0; s>>=1) {
    if ((tid < s) && (pos+s < size))
      d_in[pos] = d_in[pos] + d_in[pos+s];
    __syncthreads();
  }

  // only thread 0 writes result, as thread
  if ((tid == 0) && (pos < size))
    d_out[blockIdx.x] = d_in[pos];
}

void reduce_wrapper(unsigned int * d_out, unsigned int * d_in, int size, int num_threads) {
  int num_blocks = size / num_threads + 1;

  unsigned int * d_tmp;
  checkCudaErrors(hipMalloc(&d_tmp, sizeof(int)*num_blocks));
  checkCudaErrors(hipMemset(d_tmp, 0, sizeof(int)*num_blocks));

  int prev_num_blocks;
  int remainder = 0;
  // recursively solving, will run approximately log base num_threads times.
  do {
    reduce_kernel<<<num_blocks, num_threads>>>(d_tmp, d_in, size);

    remainder = size % num_threads;
    size = size / num_threads + remainder;

    // updating input to intermediate
    checkCudaErrors(hipMemcpy(d_in, d_tmp, sizeof(int)*num_blocks, hipMemcpyDeviceToDevice));

    // Updating num_blocks to reflect how many blocks we now want to compute on
    prev_num_blocks = num_blocks;
    num_blocks = size / num_threads + 1;      

    // updating intermediate
    checkCudaErrors(hipFree(d_tmp));
    checkCudaErrors(hipMalloc(&d_tmp, sizeof(int)*num_blocks));
  } while(size > num_threads);

  // computing rest
  reduce_kernel<<<1, size>>>(d_out, d_in, prev_num_blocks);
}

__global__
void map_kernel(unsigned int* d_out,
                unsigned int* d_in,
                unsigned int* d_predicate,
                unsigned int* d_sum_scan_0,
                unsigned int* d_sum_scan_1,
                size_t numElems) {
  int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  int pos;

  if (d_predicate[mid])
    pos = d_sum_scan_0[mid];
  else 
    pos = d_sum_scan_1[mid];

  d_out[pos] = d_in[mid];
}


void exclusive_sum_scan(unsigned int* d_out,
                        unsigned int* d_predicate,
                        unsigned int* d_predicate_tmp,
                        unsigned int* d_sum_scan,
                        unsigned int ARRAY_BYTES,
                        size_t numElems,
                        int GRID_SIZE,
                        int BLOCK_SIZE) {
  // copy predicate values to new array
  checkCudaErrors(hipMemcpy(d_predicate_tmp, d_predicate, ARRAY_BYTES, hipMemcpyDeviceToDevice));
  // set all elements to zero 
  checkCudaErrors(hipMemset(d_sum_scan, 0, ARRAY_BYTES));

  // sum scan call
  for (int step = 1; step < numElems; step *= 2) {
    inclusive_sum_scan_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_sum_scan, d_predicate_tmp, step, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpy(d_predicate_tmp, d_sum_scan, ARRAY_BYTES, hipMemcpyDeviceToDevice));
  }

  // shift to get exclusive scan
  checkCudaErrors(hipMemcpy(d_out, d_sum_scan, ARRAY_BYTES, hipMemcpyDeviceToDevice));
  right_shift_array<<<GRID_SIZE,BLOCK_SIZE>>>(d_out, d_sum_scan, numElems);
}

__global__
void toggle_predicate_kernel(unsigned int* d_out, 
                             unsigned int* d_predicate,
                             size_t numElems) {
  int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  d_out[mid] = ((d_predicate[mid]) ? 0 : 1);
}

__global__
void add_splitter_map_kernel(unsigned int* d_out,
                             unsigned int* shift, 
                             size_t numElems) {
  int mid = threadIdx.x + blockIdx.x * blockDim.x;
  if (mid >= numElems) return;

  d_out[mid] += shift[0];
}

int main(void) {
  size_t numElems = 16;
  int GRID_SIZE = 2;
  int BLOCK_SIZE = 8;
  unsigned int ARRAY_BYTES = sizeof(unsigned int) * numElems;

  // device memory
  unsigned int *d_val_src, *d_predicate, *d_sum_scan;
  checkCudaErrors(hipMalloc((void **) &d_val_src,   ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_predicate, ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_sum_scan,  ARRAY_BYTES));
  unsigned int* d_predicate_tmp;
  checkCudaErrors(hipMalloc((void **) &d_predicate_tmp, ARRAY_BYTES));

  // input array
  unsigned int* h_input = new unsigned int[numElems];
  h_input[0]  = 39; h_input[1]  = 21; h_input[2]  = 84; h_input[3]  = 40;
  h_input[4]  = 78; h_input[5]  = 85; h_input[6]  = 13; h_input[7]  = 4;
  h_input[8]  = 47; h_input[9]  = 45; h_input[10] = 91; h_input[11] = 60;
  h_input[12] = 74; h_input[13] = 8;  h_input[14] = 44; h_input[15] = 53;
  checkCudaErrors(hipMemcpy(d_val_src, h_input, ARRAY_BYTES, hipMemcpyHostToDevice));

  const unsigned int BITS_PER_BYTE = 8;

  // LOOP START
  for (unsigned int i = 0; BITS_PER_BYTE * sizeof(unsigned int); i++) {
    //##########
    // predicate call
    predicate_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_predicate, d_val_src, numElems, i);
    //##########

    //##########
    // LSB == 0
    unsigned int* d_sum_scan_0;
    checkCudaErrors(hipMalloc((void **) &d_sum_scan_0, ARRAY_BYTES));
    exclusive_sum_scan(d_sum_scan_0, d_predicate, d_predicate_tmp, d_sum_scan, ARRAY_BYTES, numElems, GRID_SIZE, BLOCK_SIZE);
    //##########

    //##########
    // reduce to get amount of LSB equal to 0
    unsigned int* d_reduce;
    checkCudaErrors(hipMalloc((void **) &d_reduce, sizeof(unsigned int)));
    // copy contents 
    checkCudaErrors(hipMemcpy(d_predicate_tmp, d_predicate, ARRAY_BYTES, hipMemcpyDeviceToDevice));
    reduce_wrapper(d_reduce, d_predicate_tmp, numElems, BLOCK_SIZE);
    unsigned int h_result;
    checkCudaErrors(hipMemcpy(&h_result, d_reduce, sizeof(int), hipMemcpyDeviceToHost));
    //##########

    //##########
    // LSB == 1
    unsigned int* d_sum_scan_1;
    unsigned int* d_predicate_toggle;
    checkCudaErrors(hipMalloc((void **) &d_sum_scan_1,       ARRAY_BYTES));
    checkCudaErrors(hipMalloc((void **) &d_predicate_toggle, ARRAY_BYTES));
    // flip predicate values
    toggle_predicate_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_predicate_toggle, d_predicate, numElems);
    exclusive_sum_scan(d_sum_scan_1, d_predicate_toggle, d_predicate_tmp, d_sum_scan, ARRAY_BYTES, numElems, GRID_SIZE, BLOCK_SIZE);
    // map sum_scan_1 to add splitter
    add_splitter_map_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_sum_scan_1, d_reduce, numElems);
    //##########

    //##########
    // move elements accordingly
    unsigned int* d_map;
    checkCudaErrors(hipMalloc((void **) &d_map, ARRAY_BYTES));
    map_kernel<<<GRID_SIZE,BLOCK_SIZE>>>(d_map, d_val_src, d_predicate, d_sum_scan_0, d_sum_scan_1, numElems);
    //##########

    // debugging
    unsigned int *h_predicate   = new unsigned int[numElems];
    unsigned int *h_predicate_toggle = new unsigned int[numElems];
    unsigned int *h_sum_scan_0  = new unsigned int[numElems];
    unsigned int *h_sum_scan_1  = new unsigned int[numElems];
    unsigned int *h_map         = new unsigned int[numElems];
    checkCudaErrors(hipMemcpy(h_predicate,   d_predicate,  ARRAY_BYTES, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_predicate_toggle, d_predicate_toggle,  ARRAY_BYTES, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_sum_scan_0,  d_sum_scan_0, ARRAY_BYTES, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_sum_scan_1,  d_sum_scan_1, ARRAY_BYTES, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_map,         d_map,        ARRAY_BYTES, hipMemcpyDeviceToHost));
   
    printf("INPUT\tPRED\tPRED_T\tSCAN_0\tSCAN_1\tMAP\n");
    for (int i = 0; i < numElems; i++)
      printf("%u\t%u\t%u\t%u\t%u\t%u\n", h_input[i], h_predicate[i], h_predicate_toggle[i], h_sum_scan_0[i], h_sum_scan_1[i], h_map[i]);

    printf("sum: \t %u\n", h_result);
  }
  // LOOP END

  return 0;
}
